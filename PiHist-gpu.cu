
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>



__global__ void frequencies(int* a, FILE* fp, int n){
    int i = 0; 
    if (i < 10) {
        a[i]++;  
    }

}

int main(int argc, char* argv[]){
    //----------------Rember to error handle--------------------
    if(argc < 3){
        printf("Not enough arguments\n");
        exit(1); 
    }
    
    // set the number of pi digits we want to count to this
    int n = atoi(argv[2]); 
    if(n < 0){
        printf("No such thing as a Negative Iterations !\n");
        exit(1);         
    }
    //open the file and check for error
    FILE * read_fp = fopen(argv[1], "r");
    if(read_fp == NULL){
        printf("The file does not exist.\n");
        exit(2);
    }


    // allocate space on cpu and gpu
    int nums[10] ={0};
    int* dev_nums;
    hipMalloc((void**) &dev_nums, 10*sizeof(float));
    // create the block and grid
    dim3 dimBlock(1024);
    dim3 dimGrid((int)ceil((float)n / 1024)); 

    
    frequencies<<<dimGrid, dimBlock>>>(dev_nums, read_fp, n);
    hipDeviceSynchronize();

    hipMemcpy(nums, dev_nums, 10*sizeof(int), hipMemcpyDeviceToHost);



    exit(0); 

}